#include "hip/hip_runtime.h"
#include <iostream>
#include "vec3.h"

#define COL 1200
#define ROW 600

__global__ void render(float *frame_buffer, int max_col, int max_row) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if ((col >= max_col) || (row >= max_row)) {
        // Pixel outside image
        return;
    }

    // Pixel index in the frame buffer (each pixel = 3 floats)
    int pixel_index = row * max_col * 3 + col * 3;
    frame_buffer[pixel_index + 0] = float(col) / max_col;
    frame_buffer[pixel_index + 1] = float(row) / max_row;
    frame_buffer[pixel_index + 2] = 0.2;
}

__host__ void write_color(std::ostream &out, color pixel_color) {
            int ir = int(255.99*pixel_color.x());
            int ig = int(255.99*pixel_color.y());
            int ib = int(255.99*pixel_color.z());
            out << ir << " " << ig << " " << ib << "\n";
} 

int main() {
    int num_pixels = COL*ROW;
    size_t frame_buffer_size = 3 * num_pixels * sizeof(float);

    // Allocate Frame Buffer
    float *frame_buffer;
    hipMallocManaged((void **)&frame_buffer, frame_buffer_size);

    // Render Frame Buffer
    int t_col = 8;
    int t_row = 8;

    // Nb of blocks in the grid
    dim3 blocks(COL/t_col + 1, ROW/t_row + 1);
    // Nb of threads in each block (one per pixel)
    dim3 threads(t_col, t_row);

    render<<<blocks, threads>>>(frame_buffer, COL, ROW);
    hipDeviceSynchronize();

    // Output frame buffer as PPM image
    std::cout << "P3\n" << COL << " " << ROW << "\n255\n";
    for(int row = ROW - 1; row >= 0; row--) {
        for(int col = 0; col < COL; col++) {
            size_t pixel_index = row*3*COL + col*3;
            color pixel_color(frame_buffer[pixel_index + 0], frame_buffer[pixel_index + 1], frame_buffer[pixel_index + 2]);
            write_color(std::cout, pixel_color);
        }
    }

    hipFree(frame_buffer);

}