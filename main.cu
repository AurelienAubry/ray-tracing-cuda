#include "hip/hip_runtime.h"
#include <iostream>
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#include "vec3.h"
#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"

#define COL 1200
#define ROW 600


__device__ float hit_sphere(const point3& center, float radius, const ray& r) {

    vec3 origin_center = r.origin() - center;
    float a = r.direction().length_squared();
    float half_b = dot(origin_center, r.direction());
    float c = origin_center.length_squared() - radius * radius;
    float discriminant = half_b*half_b - a*c;
    if(discriminant < 0) {
        return -1.0;
    } else {
        return (-half_b -sqrt(discriminant)) / a;
    }
}

__device__ color ray_color(const ray& r, hittable **world) {
    color white = color(1.0, 1.0, 1.0);
    color blue = color(0.5, 0.7, 1.0);
    color red = color(1.0, 0.0, 0.0);

    hit_record rec;
    if((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5 * (rec.normal + color(1,1,1));
    }
    
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5*(unit_direction.y() + 1.0);
    return (1.0-t)*white + t*blue;
}

__global__ void render_init(int max_row, int max_col, hiprandState *rand_state) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if ((col >= max_col) || (row >= max_row)) {
        // Pixel outside image
        return;
    }

    int pixel_index = row * max_col + col;

    // Initialize random number generator for the current thread
    hiprand_init(2021, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(color *frame_buffer, int max_col, int max_row, camera **cam, hittable **world, hiprandState *rand_state, int samples_per_pixel) {

    point3 origin = point3(0, 0, 0);
    point3 lower_left_corner = point3(-2.0, -1.0, -1.0);
    vec3 horizontal = vec3(4.0, 0.0, 0);
    vec3 vertical = vec3(0.0, 2.0, 0);

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if ((col >= max_col) || (row >= max_row)) {
        // Pixel outside image
        return;
    }

    // Pixel index in the frame buffer (each pixel = 3 floats)
    int pixel_index = row * max_col + col;

    hiprandState local_rand_state = rand_state[pixel_index];
    color pixel_color(0, 0, 0);
    for(int s = 0; s < samples_per_pixel; s++) {
        float u = float(col + hiprand_uniform(&local_rand_state)) / float(max_col);
        float v = float(row + hiprand_uniform(&local_rand_state)) / float(max_row);
        ray r(origin, lower_left_corner + u*horizontal + v*vertical);
        pixel_color += ray_color(r, world);
    }
    
    frame_buffer[pixel_index] = pixel_color;
}

__global__ void create_world(hittable **d_objects_list, hittable **d_world, camera **d_camera) {
    // Make sure this is only executed once
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_objects_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_objects_list+1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_objects_list, 2);
        *d_camera   = new camera();
    }
}

__global__ void free_world(hittable **d_objects_list, hittable **d_world, camera **d_camera) {
   delete *(d_objects_list);
   delete *(d_objects_list+1);
   delete *d_world;
   delete *d_camera;
}

__host__ inline float clamp(float x, float min, float max) {
    if (x < min) return min;
    if (x > max) return max;
    return x;
}

__host__ void write_color(std::ostream &out, color pixel_color, int samples_per_pixel) {
            float r = pixel_color.x();
            float g = pixel_color.y();
            float b = pixel_color.z();

            float scale = 1.0 / samples_per_pixel;
            r *= scale;
            g *= scale;
            b *= scale;

            int ir = int(255.99*clamp(r, 0.0, 0.999));
            int ig = int(255.99*clamp(g, 0.0, 0.999));
            int ib = int(255.99*clamp(b, 0.0, 0.999));
            out << ir << " " << ig << " " << ib << "\n";
} 

int main() {
    const int num_pixels = COL*ROW;
    const int samples_per_pixel = 100;
    size_t frame_buffer_size = num_pixels * sizeof(color);

    // Allocate Frame Buffer
    color *frame_buffer;
    hipMallocManaged((void **)&frame_buffer, frame_buffer_size);

    // Allocate world
    hittable **d_objects_list;
    hipMalloc((void **)&d_objects_list, 2*sizeof(hittable *));
    hittable **d_world;
    hipMalloc((void **)&d_world, sizeof(hittable *));
    camera **d_camera;
    hipMalloc((void **)&d_camera, sizeof(camera *));
    create_world<<<1,1>>>(d_objects_list, d_world, d_camera);
    hipDeviceSynchronize();


    // Render Frame Buffer
    int t_col = 8;
    int t_row = 8;

    // Nb of blocks in the grid
    dim3 blocks(COL/t_col + 1, ROW/t_row + 1);
    // Nb of threads in each block (one per pixel)
    dim3 threads(t_col, t_row);

    // List of pixels random number generator states
    hiprandState *d_rand_state;
    hipMalloc((void**)&d_rand_state, num_pixels*sizeof(hiprandState));

    render_init<<<blocks, threads>>>(COL, ROW, d_rand_state);
    hipDeviceSynchronize();

    render<<<blocks, threads>>>(frame_buffer, COL, ROW, d_camera, d_world, d_rand_state, samples_per_pixel);
    hipDeviceSynchronize();

    // Output frame buffer as PPM image
    std::cout << "P3\n" << COL << " " << ROW << "\n255\n";
    for(int row = ROW - 1; row >= 0; row--) {
        for(int col = 0; col < COL; col++) {
            size_t pixel_index = row*COL + col;
            write_color(std::cout, frame_buffer[pixel_index], samples_per_pixel);
        }
    }

    // Clean up
    hipDeviceSynchronize();
    free_world<<<1,1>>>(d_objects_list, d_world, d_camera);
    hipFree(d_objects_list);
    hipFree(d_world);
    hipFree(frame_buffer);

    hipDeviceReset();
}