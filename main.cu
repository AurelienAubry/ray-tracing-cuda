#include "hip/hip_runtime.h"
#include <iostream>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"

#define COL 1200
#define ROW 600


__device__ float hit_sphere(const point3& center, float radius, const ray& r) {

    vec3 origin_center = r.origin() - center;
    float a = r.direction().length_squared();
    float half_b = dot(origin_center, r.direction());
    float c = origin_center.length_squared() - radius * radius;
    float discriminant = half_b*half_b - a*c;
    if(discriminant < 0) {
        return -1.0;
    } else {
        return (-half_b -sqrt(discriminant)) / a;
    }
}

__device__ color ray_color(const ray& r, hittable **world) {
    color white = color(1.0, 1.0, 1.0);
    color blue = color(0.5, 0.7, 1.0);
    color red = color(1.0, 0.0, 0.0);

    hit_record rec;
    if((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5 * (rec.normal + color(1,1,1));
    }
    
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5*(unit_direction.y() + 1.0);
    return (1.0-t)*white + t*blue;
}


__global__ void render(color *frame_buffer, int max_col, int max_row, hittable **world) {

    point3 origin = point3(0, 0, 0);
    point3 lower_left_corner = point3(-2.0, -1.0, -1.0);
    vec3 horizontal = vec3(4.0, 0.0, 0);
    vec3 vertical = vec3(0.0, 2.0, 0);

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if ((col >= max_col) || (row >= max_row)) {
        // Pixel outside image
        return;
    }

    // Pixel index in the frame buffer (each pixel = 3 floats)
    int pixel_index = row * max_col + col;
    float u = float(col) / float(max_col);
    float v = float(row) / float(max_row);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    frame_buffer[pixel_index] = ray_color(r, world);
}

__global__ void create_world(hittable **d_objects_list, hittable **d_world) {
    // Make sure this is only executed once
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_objects_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_objects_list+1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_objects_list, 2);
    }
}

__global__ void free_world(hittable **d_objects_list, hittable **d_world) {
   delete *(d_objects_list);
   delete *(d_objects_list+1);
   delete *d_world;
}

__host__ void write_color(std::ostream &out, color pixel_color) {
            int ir = int(255.99*pixel_color.x());
            int ig = int(255.99*pixel_color.y());
            int ib = int(255.99*pixel_color.z());
            out << ir << " " << ig << " " << ib << "\n";
} 

int main() {
    int num_pixels = COL*ROW;
    size_t frame_buffer_size = num_pixels * sizeof(color);

    // Allocate Frame Buffer
    color *frame_buffer;
    hipMallocManaged((void **)&frame_buffer, frame_buffer_size);

    // Allocate world
    hittable **d_objects_list;
    hipMalloc((void **)&d_objects_list, 2*sizeof(hittable *));
    hittable **d_world;
    hipMalloc((void **)&d_world, sizeof(hittable *));
    create_world<<<1,1>>>(d_objects_list,d_world);
    hipDeviceSynchronize();


    // Render Frame Buffer
    int t_col = 8;
    int t_row = 8;

    // Nb of blocks in the grid
    dim3 blocks(COL/t_col + 1, ROW/t_row + 1);
    // Nb of threads in each block (one per pixel)
    dim3 threads(t_col, t_row);

    render<<<blocks, threads>>>(frame_buffer, COL, ROW, d_world);
    hipDeviceSynchronize();

    // Output frame buffer as PPM image
    std::cout << "P3\n" << COL << " " << ROW << "\n255\n";
    for(int row = ROW - 1; row >= 0; row--) {
        for(int col = 0; col < COL; col++) {
            size_t pixel_index = row*COL + col;
            write_color(std::cout, frame_buffer[pixel_index]);
        }
    }

    // Clean up
    hipDeviceSynchronize();
    free_world<<<1,1>>>(d_objects_list, d_world);
    hipFree(d_objects_list);
    hipFree(d_world);
    hipFree(frame_buffer);

    hipDeviceReset();
}